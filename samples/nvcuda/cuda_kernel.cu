#include "hip/hip_runtime.h"
/*
 * Tuan Le
 * University of California, Berkeley
 * Berkeley Space Sciences Lab
 * tuanle86@berkeley.edu
 */

// When VERIFY is defined, the sum of squared errors is calculated between the
// identity matrix and the product A * incerse(A). For debugging...
//#define VERIFY 1

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "config.h"

void mathdispAI(const REAL *mat, int lda, int MAT_SIZE_h) {
    fprintf(stderr, "\n");
	int i,j;
    for (j=0;j<MAT_SIZE_h;j++)  {
		for (i=0;i<MAT_SIZE_h*2;i++) {
	    	fprintf(stderr, "%6.3f",mat[j*lda*2+i]);
	}
	fprintf(stderr, "\n");
    }
    fprintf(stderr, "\n");
} // mathdisp2

void mathdispAId(const REAL * AId, int lda, int n) {
	REAL * AI = (REAL *)malloc(sizeof(REAL)*(n*lda*2));
	hipMemcpy(AI,AId,sizeof(REAL)*n*lda*2,hipMemcpyDeviceToHost);
	mathdispAI(AI, lda, n);
	delete [] AI;
}

__global__ void GEStep1A(REAL * AI, int i, int n2, int lda2) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;

  if (k>i && k < n2 && AI[i*lda2+k]!=0) {
    REAL multiplyer = -AI[i*lda2+k]/AI[i*lda2+i];
	int n = n2 / 2;
    for (int j = i+1; j < n; j++) {
      AI[j*lda2+k] += multiplyer*AI[j*lda2+i];
    }
  }
}

__global__ void GEStep2(REAL * AI,REAL diag,int i, int n2, int lda2) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	if (k < n2) {
		AI[i*lda2+k] /= diag;
	}
}

__global__ void GEStep3(REAL * AI,int i, int n2, int lda2) {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k > i && k < n2) {
    REAL multiplyer = -AI[i*lda2+k];
    for (int j = 0; j < i; j++) {
      AI[j*lda2+k] += multiplyer*AI[j*lda2+i];
    }
  }
}

//extern void invert(REAL * A, int n);
void invertge(REAL * AI_d, int lda, int n) {
	int lda2 = lda * 2;
  // perform elementary row operations till A in AI becomes identity matrix
  for (int i = 0; i < n; i++) {
    GEStep1A<<<(int)ceil((float)(1+(2*n-1)/32)),32>>>(AI_d,i,n*2, lda2);
    CUDACHECK;
    hipDeviceSynchronize();
  }


  for (int i = n-1; i >= 0; i--) {
    REAL diag = 1.0;
    SAFECALL(hipMemcpy(&diag, &AI_d[i*lda2+i], sizeof(REAL), hipMemcpyDeviceToHost));
    GEStep2<<<(int)ceil((float)(1+(n*2-1)/32)),32>>>(AI_d,diag,i,n*2, lda2);
    CUDACHECK;

    GEStep3<<<(int)ceil((float)(1+(n*2-1)/32)),32>>>(AI_d,i,n*2, lda2);
    CUDACHECK;
    hipDeviceSynchronize();
    CUDACHECK;
  }
} // invertge


/* inverts nxn matrix A and stores result back in A */
void invert(REAL * A, int n) {
fprintf(stderr,"starting inversion n = %d ", n);
    volatile clock_t gputime, gputime0;
    gputime=clock();
    gputime0 = gputime;

    int lda = ((n+15)&~15|16);
//lda=n;
	REAL * AI = (REAL *)malloc(sizeof(REAL)*(n*lda*2));
	memset(AI,0,sizeof(REAL)*n*lda*2);
	for (int i = 0; i < n; i++) {
		memcpy(&AI[lda*i*2], &A[n*i], sizeof(REAL)*n);
		AI[lda*i*2+n+i] = 1;
	}

	REAL * AI_d;
	SAFECALL(hipMalloc((void **) &AI_d, sizeof(REAL)*n*lda*2));
	SAFECALL(hipMemcpy(AI_d, AI, sizeof(REAL)*n*lda*2, hipMemcpyHostToDevice));

	invertge(AI_d, lda, n);
	SAFECALL(hipMemcpy(AI, AI_d, sizeof(REAL)*n*lda*2, hipMemcpyDeviceToHost));
	hipFree(AI_d);


	gputime=clock()-gputime;fprintf(stderr, " %7.1f ms ",gputime/1.e3f);
    	fprintf(stderr, " %7.2f Gflops", 1e-3*(3.0)*n*n*n/3.0/gputime);
#ifdef VERIFY	
	// let's verify that
	REAL error=0.0;

	// multiply inverse*xcopy, should be Identity matrix
	for (int k = 0; k < n; k++) {
	  for (int j = 0; j < n; j++) {
	    REAL sum = 0;
	    for (int i = 0; i < n; i++) {
	      sum += AI[j*lda*2+n+i]*A[i*n+k];
	    }
	    if (j!=k) {
	      error += sum * sum;
	    } else {
	      error += (1.0-sum) * (1.0-sum);
	    }
	  }
	}
    fprintf(stderr, " %6.2f SSE", error);
#endif	

	for (int i = 0; i < n; i++) {
		memcpy(&A[n*i], &AI[lda*i*2+n], sizeof(REAL)*n);
	}
	free(AI);
	fprintf(stderr," done!\n");
} // invert
